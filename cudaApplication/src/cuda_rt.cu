#include "hip/hip_runtime.h"
#include "cuda_rt.cuh"
#include "shapes/triangle.cuh"
#include "utils/ray.cuh"
#include "shapes/mesh.cuh"

__global__ void d_raytrace(Ray* rays, Triangle* triangles, int num_triangles, double* output, Eigen::Vector3d* light_positions, int num_lights) {
    int idx = threadIdx.x * blockDim.x + blockIdx.x;
    Ray r = rays[idx];

    double min_t = 1000000.0;
    int min_index = -1;
    for (int i = 0; i < num_triangles; i++) {
        double t = triangles[i].intersects(r);
        if (t > 0 && t < min_t) {
            min_t = t;
            min_index = i;
        }
    }

    Eigen::Vector3d p = r.origin + r.direction*min_t;

    Triangle closest = triangles[min_index];
    auto N = closest.normal();
    double brightness = 0.;
    for (int i = 0; i < num_lights; i++) {
        auto L = (light_positions[i] - p).normalized();
        double lambertian = N.dot(L);
        lambertian = lambertian < 0 ? 0 : lambertian;
        brightness += lambertian;
    }

    output[idx] = brightness;

}

double* h_raytrace(Ray* rays, Mesh mesh, int width, int height, std::vector<Eigen::Vector3d> light_positions) {
    int size = width*height;
    int num_triangles = mesh.triangles.size();
    int num_lights = light_positions.size();

    double* h_output = new double[size];
    
    Ray* d_rays = nullptr;
    Triangle* d_triangles = nullptr;
    double* d_output = nullptr;
    Eigen::Vector3d* d_lights = nullptr;
    
    hipMalloc((void**)&d_rays, size*sizeof(Ray));
    hipMalloc((void**)&d_triangles, num_triangles*sizeof(Triangle));
    hipMalloc((void**)&d_output, size*sizeof(double));
    hipMalloc((void**)&d_lights, num_lights*sizeof(Eigen::Vector3d));

    hipMemcpy(d_rays, rays, size*sizeof(Ray), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, &mesh.triangles[0], num_triangles*sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, &light_positions[0], num_lights*sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);

    d_raytrace<<<width, height>>>(d_rays, d_triangles, num_triangles, d_output, d_lights, num_lights);

    hipMemcpy(h_output, d_output, size*sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_rays);
    hipFree(d_triangles);
    hipFree(d_output);
    
    return h_output;
}