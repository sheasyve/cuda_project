#include "hip/hip_runtime.h"
#include "cuda_rt.cuh"
#include "shapes/triangle.cuh"
#include "utils/ray.cuh"
#include "shapes/mesh.cuh"

__global__ void d_raytrace(
    Ray* rays, Triangle* triangles,
    int num_triangles, double* output,
    Eigen::Vector3d* light_positions,
    Eigen::Vector4d* light_colors, int num_lights
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int width = gridDim.x;
    int idx = y * width + x;//Get index from pixel x and y
    if (idx >= width * gridDim.y) return;//Grid bounds check

    Ray r = rays[idx];
    double min_t = 1e20;
    int min_index = -1;

    for (int i = 0; i < num_triangles; i++) {//Closest Intersection
        double t = triangles[i].intersects(r);
        if (t > 0 && t < min_t) {
            min_t = t;
            min_index = i;
        }
    }
    if (min_index == -1) {
        output[idx] = 0.0;
        return;
    }

    Eigen::Vector3d p = r.origin + r.direction * min_t;
    Triangle closest = triangles[min_index];
    Eigen::Vector3d N = closest.normal();
    N.normalize();
    Eigen::Vector3d V = -r.direction;
    V.normalize();

    double brightness = 0.005;//Start with ambient light
    double diffuse_intensity = 0.4;
    double specular_intensity = 0.4;
    double shine = 32.0;
  
    for (int i = 0; i < num_lights; i++) {//Add light from each source
        Eigen::Vector3d L = (light_positions[i] - p);
        double d = L.norm();
        L.normalize();
        Eigen::Vector3d light_rgb = light_colors[i].head<3>();
        double a=1.,b=.1,c=.01;
        double attenuation = 1./(a+b*d+c*d*d);//Distance dropoff
        //Diffuse
        double lambertian = N.dot(L);
        lambertian = fmax(lambertian, 0.0);
        brightness += attenuation * diffuse_intensity * lambertian * light_rgb.norm();
        //Specular
        Eigen::Vector3d R = (2.0 * N.dot(L) * N - L).normalized();
        double spec_angle = R.dot(V);
        spec_angle = fmax(spec_angle, 0.0);
        double specular = pow(spec_angle, shine);
        brightness += attenuation * specular_intensity * specular * light_rgb.norm();
        brightness = fmin(brightness, 1.0);
    }
    output[idx] = fmin(brightness, 1.0);
}

std::vector<Triangle> collect_all_triangles(const std::vector<Mesh>& meshes) {
    std::vector<Triangle> triangles;
    for (const auto& mesh : meshes) {
        triangles.insert(triangles.end(), mesh.triangles.begin(), mesh.triangles.end());
    }
    return triangles;
}

double* h_raytrace(
    Ray* rays, std::vector<Mesh> meshes,
    int width, int height,
    std::vector<Eigen::Vector3d> light_positions,
    std::vector<Eigen::Vector4d> light_colors
) {
    int size = width * height;
    int num_lights = light_positions.size();

    std::vector<Triangle> triangles = collect_all_triangles(meshes);
    int num_triangles = triangles.size();

    double* h_output = new double[size];

    Ray* d_rays = nullptr;
    Triangle* d_triangles = nullptr;
    double* d_output = nullptr;
    Eigen::Vector3d* d_lights = nullptr;
    Eigen::Vector4d* d_light_colors = nullptr;

    hipMalloc((void**)&d_rays, size * sizeof(Ray));
    hipMalloc((void**)&d_triangles, num_triangles * sizeof(Triangle));
    hipMalloc((void**)&d_output, size * sizeof(double));
    hipMalloc((void**)&d_lights, num_lights * sizeof(Eigen::Vector3d));
    hipMalloc((void**)&d_light_colors, num_lights * sizeof(Eigen::Vector4d));

    hipMemcpy(d_rays, rays, size * sizeof(Ray), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, triangles.data(), num_triangles * sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, light_positions.data(), num_lights * sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(d_light_colors, light_colors.data(), num_lights * sizeof(Eigen::Vector4d), hipMemcpyHostToDevice);

    dim3 gridDim(width, height);
    d_raytrace<<<gridDim, 1>>>(d_rays, d_triangles, num_triangles, d_output, d_lights, d_light_colors, num_lights);

    hipMemcpy(h_output, d_output, size * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_rays);
    hipFree(d_triangles);
    hipFree(d_output);
    hipFree(d_lights);
    hipFree(d_light_colors);

    return h_output;
}
