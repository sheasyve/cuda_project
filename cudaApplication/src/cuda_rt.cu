#include "hip/hip_runtime.h"
#include "cuda_rt.cuh"

__global__ void d_raytrace(
    Ray* rays, BvhTree::Node* nodes, int root_index, Triangle* triangles,
    double* output,
    int width, int height,
    Eigen::Vector3d* light_positions,
    Eigen::Vector4d* light_colors, int num_lights
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;//Ray index for this thread
    Ray r = rays[idx];
    double min_t = INF;
    int mindex = find_closest_triangle(r, nodes, root_index, triangles, min_t);//Done in cuda_rt_util.cu by traversing the BVH tree

    if (mindex == -1) {
        output[idx] = 0.0;
        return;
    }

    Eigen::Vector3d p = r.origin + r.direction * min_t;
    Triangle closest = triangles[mindex];
    Eigen::Vector3d N = closest.normal();
    N.normalize();
    Eigen::Vector3d V = -r.direction;
    V.normalize();

    double brightness = 0.005; // Start with ambient light
    double diffuse_intensity = 0.4;
    double specular_intensity = 0.4;
    double shine = 32.0;

    for (int i = 0; i < num_lights; i++) { // Add light from each source
        Eigen::Vector3d L = (light_positions[i] - p);
        double d = L.norm();
        L.normalize();
        Eigen::Vector3d light_rgb = light_colors[i].head<3>();
        double a = 1.0, b = 0.1, c = 0.01;
        double attenuation = 1.0 / (a + b * d + c * d * d); // Distance dropoff
        // Diffuse
        double lambertian = N.dot(L);
        lambertian = fmax(lambertian, 0.0);
        brightness += attenuation * diffuse_intensity * lambertian * light_rgb.norm();
        // Specular
        Eigen::Vector3d R = (2.0 * N.dot(L) * N - L).normalized();
        double spec_angle = R.dot(V);
        spec_angle = fmax(spec_angle, 0.0);
        double specular = pow(spec_angle, shine);
        brightness += attenuation * specular_intensity * specular * light_rgb.norm();
    }
    brightness = fmin(brightness, 1.0);
    output[idx] = brightness;
}

double* h_raytrace(
    Ray* rays, std::vector<Mesh> meshes,
    int width, int height,
    std::vector<Eigen::Vector3d> light_positions,
    std::vector<Eigen::Vector4d> light_colors
) {
    int size = width * height;
    int num_lights = static_cast<int>(light_positions.size());

    std::vector<Triangle> triangles = get_triangles(meshes);
    int num_triangles = static_cast<int>(triangles.size());

    BvhTree bvh(triangles);//Build BVH from triangles
    std::vector<BvhTree::Node>& nodes = bvh.nodes;//Nodes of bvh tree
    int tree_size = static_cast<int>(nodes.size());
    int root = bvh.root;

    double* h_output = new double[size];

    // Device pointers
    Ray* d_rays = nullptr;
    Triangle* d_triangles = nullptr;
    BvhTree::Node* d_nodes = nullptr;
    double* d_output = nullptr;
    Eigen::Vector3d* d_lights = nullptr;
    Eigen::Vector4d* d_light_colors = nullptr;

    hipMalloc((void**)&d_rays, size * sizeof(Ray));
    hipMalloc((void**)&d_triangles, num_triangles * sizeof(Triangle));
    hipMalloc((void**)&d_nodes, tree_size * sizeof(BvhTree::Node));
    hipMalloc((void**)&d_output, size * sizeof(double));
    hipMalloc((void**)&d_lights, num_lights * sizeof(Eigen::Vector3d));
    hipMalloc((void**)&d_light_colors, num_lights * sizeof(Eigen::Vector4d));

    hipMemcpy(d_rays, rays, size * sizeof(Ray), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, triangles.data(), num_triangles * sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes, nodes.data(), tree_size * sizeof(BvhTree::Node), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, light_positions.data(), num_lights * sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(d_light_colors, light_colors.data(), num_lights * sizeof(Eigen::Vector4d), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,(height + blockDim.y - 1) / blockDim.y);

    d_raytrace<<<gridDim, blockDim>>>(
        d_rays, d_nodes, root, d_triangles,
        d_output,
        width, height,
        d_lights, d_light_colors, num_lights
    );

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("CUDA error. %s\n", hipGetErrorString(err));
    
    hipMemcpy(h_output, d_output, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_rays);
    hipFree(d_triangles);
    hipFree(d_nodes);
    hipFree(d_output);
    hipFree(d_lights);
    hipFree(d_light_colors);

    return h_output;
}

