#include "hip/hip_runtime.h"
#include "utils/main_util.cuh"
#include "cuda_rt.cuh"

// Camera settings
const double focal_length = 2.16;
const double field_of_view = 0.7854; // 45 degrees
const Eigen::Vector3d camera_position(0, 0, -100);

// Lights
std::vector<Eigen::Vector3d> light_positions;
std::vector<Eigen::Vector4d> light_colors;

// Variant to store different objects
using Intersectable = std::variant<Triangle, Sphere, Mesh>;
std::vector<Intersectable> objects;

Eigen::Vector3d compute_normal(const std::variant<Triangle, Sphere, Mesh> &obj, const Eigen::Vector3d &hit_point, const Triangle *hit_triangle = nullptr){
    return std::visit([&](const auto &shape) -> Eigen::Vector3d{
        if constexpr (std::is_same_v<decltype(shape), const Sphere&>) {//Sphere
            return (hit_point - shape.center).normalized();
        } else if constexpr (std::is_same_v<decltype(shape), const Triangle&>) {//Triangle
            return shape.normal();
        } else if constexpr (std::is_same_v<decltype(shape), const Mesh&>) {//Mesh
            if (hit_triangle != nullptr) {
                return hit_triangle->normal();  
            }
            return Eigen::Vector3d(0, 0, 0);  // No valid hit triangle, fallback
        }
    }, obj);
}

void print_scene_in_ascii(double* color, int w, int h) {
    // ASCII characters for brightness levels
    const std::string brightness_chars = " `.-':_,^=;><+!rc*/z?sLTv)J7(|Fi{C}fI31tlu[neoZ5Yxjya]2ESwqkP6h9d4VpOGbUAKXHm8RD#$Bg0MNWQ%&@";
    const int l = brightness_chars.size() - 1;
    auto [first_line, last_line] = find_boundary(color, w, h);
    for (int j = first_line; j >= last_line; --j) {
        for (int i = 0; i < w; ++i) {
            double brightness = color[j*w + i];
            brightness = std::max(0.0, std::min(1.0, brightness)); // Clamp brightness between 0 and 1
            char c = brightness_chars[static_cast<int>(l * brightness)];
            std::cout << c;
        }
        std::cout << std::endl;
    }
}

void setup_scene(){ 
    light_colors.emplace_back(0.8, 0.8, 0.8, 1);  
    light_positions.emplace_back(0, 5, -30);  

    light_colors.emplace_back(0.4, 0.4, 0.4, 1); 
    light_positions.emplace_back(10, -5, -20);  

    light_colors.emplace_back(0.3, 0.3, 0.3, 1);  
    light_positions.emplace_back(10, 5, 20);  

    light_colors.emplace_back(0.2, 0.2, 0.2, 1);  
    light_positions.emplace_back(-10, 20, -30);  
}

std::vector<Ray> gen_rays(int w, int h) {
    std::vector<Ray> rays;
    const double aspect_ratio = double(w) / double(h);
    const double y = (((focal_length)*sin(field_of_view / 2)) / sin((180 - (90 + ((field_of_view * (180 / M_PI) / 2)))) * (M_PI / 180)));
    const double x = (y * aspect_ratio);
    Eigen::Vector3d image_origin(-x, y, camera_position[2] - focal_length);
    Eigen::Vector3d x_displacement(2.0 / w * x, 0, 0);
    Eigen::Vector3d y_displacement(0, -2.0 / h * y, 0);
    for (int j = 0; j < h; j++) {
        for (int i = 0; i < w; i++) {
            Eigen::Vector3d pixel_center = image_origin + (i + 0.5) * x_displacement + (j + 0.5) * y_displacement;
            Ray r(camera_position, (camera_position - pixel_center).normalized());
            rays.push_back(r);
        }
    }
    return rays;
}

int main(int argc, char* argv[]){
    std::istream *input_stream = nullptr;
    std::ifstream file_stream;
    if (argc >= 2) {
        std::string obj_file_path = argv[1];
        file_stream.open(obj_file_path);
        if (!file_stream) {
            std::cerr << "can't open file" << std::endl;
            return 1;
        }
        input_stream = &file_stream; 
    } else {
        input_stream = &std::cin;  
    }
    
    setup_scene();
    LoadMesh m(Eigen::Matrix4d::Identity(), *input_stream);
    Mesh mesh = m.get_mesh();
    int w = 112*2, h = 224*2;
    
    //Rotation IN RADIANS
    double rX =-.05, rY =.4, rZ =.05;
    double* output = h_raytrace(&gen_rays(w, h)[0], mesh, w, h, light_positions,light_colors,rX,rY,rZ);//Cuda Kernel
    print_scene_in_ascii(output, w, h);
    return 0;
}
